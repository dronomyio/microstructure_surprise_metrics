#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"
#include <cstdio>
#include <cmath>

namespace surprise_metrics {
namespace cuda {

// Optimized GARCH kernel using shared memory
__global__ void garch_kernel(
    const float* __restrict__ returns,
    float* __restrict__ sigma_squared,
    const int n,
    const float omega,
    const float alpha,
    const float beta
) {
    extern __shared__ float shared_mem[];
    
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;
    
    // Load returns to shared memory with coalesced access
    if (gid < n) {
        shared_mem[tid] = returns[gid] * returns[gid];
    }
    __syncthreads();
    
    // Initialize first value
    if (gid == 0) {
        sigma_squared[0] = omega / (1.0f - alpha - beta);
    }
    __syncthreads();
    
    // Sequential GARCH update within blocks
    if (gid > 0 && gid < n) {
        float prev_sigma2 = (gid > 1) ? sigma_squared[gid-1] : sigma_squared[0];
        float curr_sigma2 = omega + alpha * shared_mem[tid-1] + beta * prev_sigma2;
        sigma_squared[gid] = curr_sigma2;
    }
}

// Lee-Mykland kernel with warp-level primitives
__global__ void lee_mykland_kernel(
    const float* __restrict__ returns,
    float* __restrict__ local_vol,
    float* __restrict__ test_stats,
    bool* __restrict__ jump_flags,
    const int n,
    const int window_size,
    const float threshold
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (gid >= window_size && gid < n) {
        // Compute bipower variation in window
        float bv = 0.0f;
        const float pi_over_2 = 1.5707963267948966f;
        
        #pragma unroll 4
        for (int i = gid - window_size + 1; i < gid; i++) {
            bv += fabsf(returns[i]) * fabsf(returns[i-1]);
        }
        bv *= pi_over_2 / (window_size - 1);
        
        local_vol[gid] = sqrtf(bv);
        
        // Compute test statistic
        float L = fabsf(returns[gid]) / local_vol[gid];
        test_stats[gid] = L;
        
        // Jump detection with threshold
        float Cn = sqrtf(2.0f * logf(float(n)));
        float Sn = 1.0f/Cn + (logf(3.14159f) + logf(2.0f * logf(float(n)))) / (2.0f * Cn);
        float critical_value = threshold + Cn * Sn;
        
        jump_flags[gid] = (L > critical_value);
    }
}

// BNS kernel
__global__ void bns_kernel(
    const float* __restrict__ returns,
    float* __restrict__ rv,
    float* __restrict__ bv,
    float* __restrict__ tq,
    float* __restrict__ test_stats,
    const int n,
    const int window_size
) {
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;
    
    if (gid < n - window_size) {
        float local_rv = 0.0f;
        float local_bv = 0.0f;
        float local_tq = 0.0f;
        
        const float pi_over_2 = 1.5707963267948966f;
        const float mu_43 = 1.67f; // E[|Z|^(4/3)] for standard normal
        
        // Compute statistics over window
        for (int i = 0; i < window_size; i++) {
            int idx = gid + i;
            float r = returns[idx];
            local_rv += r * r;
            
            if (i > 0) {
                local_bv += fabsf(r) * fabsf(returns[idx-1]);
            }
            
            if (i > 1) {
                float r1 = powf(fabsf(returns[idx]), 4.0f/3.0f);
                float r2 = powf(fabsf(returns[idx-1]), 4.0f/3.0f);
                float r3 = powf(fabsf(returns[idx-2]), 4.0f/3.0f);
                local_tq += r1 * r2 * r3;
            }
        }
        
        local_bv *= pi_over_2 * window_size / (window_size - 1);
        //local_tq *= window_size * powf(mu_43, -3.0f) * window_size / (window_size - 2);
	//local_tq *= (window_size / (window_size - 2)) * powf(mu_43, -3.0f);
	local_tq *= powf(mu_43, -3.0f) * window_size / (window_size - 2);
        
        // Store results
        rv[gid] = local_rv;
        bv[gid] = local_bv;
        tq[gid] = local_tq;
	 
	//ref https://public.econ.duke.edu/~get/browse/courses/883/Spr15/COURSE-MATERIALS/Z_Papers/BNSJFEC2006.pdf
	//page 8 equ 11
        
        // Compute BNS test statistic
        float jump_component = local_rv - local_bv; //RV - BV
        float theta = (pi_over_2 * pi_over_2 / 4.0f + 3.14159f - 5.0f); //ϑ
        float denominator = sqrtf(theta * local_tq / (local_bv * local_bv));
        
        test_stats[gid] = sqrtf(float(window_size)) * (jump_component / local_rv) / denominator;
    }
}

// Hawkes intensity kernel with exponential decay
__global__ void hawkes_intensity_kernel(
    const float* __restrict__ timestamps,
    float* __restrict__ intensity,
    const int n,
    const float mu,
    const float phi,
    const float kappa,
    const float dt
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (gid < n) {
        float lambda = mu;
        float t_current = timestamps[gid];
        
        // Sum contributions from all previous events
        for (int j = 0; j < gid; j++) {
            float time_diff = (t_current - timestamps[j]) * 1e-9f; // Convert ns to seconds
            if (time_diff > 0) {
                lambda += phi * expf(-kappa * time_diff);
            }
        }
        
        intensity[gid] = lambda;
    }
}

// Standardized returns kernel using vectorized operations
__global__ void standardized_returns_kernel(
    const float* __restrict__ returns,
    const float* __restrict__ sigma,
    float* __restrict__ z_scores,
    const int n
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Grid-stride loop for better GPU utilization
    for (int i = gid; i < n; i += stride) {
        z_scores[i] = returns[i] / sigma[i];
    }
}

// Host launcher functions
void launch_garch_estimation(float* returns, float* sigma, int n, 
                             float omega, float alpha, float beta,
                             hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    size_t shared_mem_size = BLOCK_SIZE * sizeof(float);
    
    garch_kernel<<<blocks, BLOCK_SIZE, shared_mem_size, stream>>>(
        returns, sigma, n, omega, alpha, beta
    );
}

void launch_jump_detection(float* returns, float* local_vol, 
                           bool* jump_flags, int n, float threshold,
                           hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // Allocate test_stats on device
    float* test_stats;
    hipMalloc(&test_stats, n * sizeof(float));
    
    lee_mykland_kernel<<<blocks, BLOCK_SIZE, 0, stream>>>(
        returns, local_vol, test_stats, jump_flags, n, 100, threshold
    );
    
    hipFree(test_stats);
}

void launch_hawkes_intensity(float* timestamps, float* intensity, 
                             int n, float mu, float phi, float kappa,
                             hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    hawkes_intensity_kernel<<<blocks, BLOCK_SIZE, 0, stream>>>(
        timestamps, intensity, n, mu, phi, kappa, 0.001f
    );
}

}} // namespace surprise_metrics::cuda
