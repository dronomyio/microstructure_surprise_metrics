#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"
#include <cstdio>
#include <cmath>
#include <float.h>

namespace surprise_metrics {
namespace cuda {

// ==================== KERNEL IMPLEMENTATIONS ====================

// Optimized GARCH kernel using shared memory
__global__ void garch_kernel(
    const float* __restrict__ returns,
    float* __restrict__ sigma_squared,
    const int n,
    const float omega,
    const float alpha,
    const float beta
) {
    extern __shared__ float shared_mem[];
    
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;
    
    if (gid < n) {
        shared_mem[tid] = returns[gid] * returns[gid];
    }
    __syncthreads();
    
    if (gid == 0) {
        sigma_squared[0] = omega / (1.0f - alpha - beta);
    }
    __syncthreads();
    
    if (gid > 0 && gid < n) {
        float prev_sigma2 = (gid > 1) ? sigma_squared[gid-1] : sigma_squared[0];
        float curr_sigma2 = omega + alpha * shared_mem[tid-1] + beta * prev_sigma2;
        sigma_squared[gid] = curr_sigma2;
    }
}

// Lee-Mykland kernel with corrected bipower variation
__global__ void lee_mykland_kernel(
    const float* __restrict__ returns,
    float* __restrict__ local_vol,
    float* __restrict__ test_stats,
    char* __restrict__ jump_flags,  // Changed bool* to char*
    const int n,
    const int window_size,
    const float threshold
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (gid >= window_size && gid < n) {
        float bv = 0.0f;
        const float pi_over_2 = 1.5707963267948966f;
        
        // Fix: Ensure we don't access returns[i-1] when i=0
        int start_idx = (gid - window_size + 1 > 1) ? (gid - window_size + 1) : 1;
        for (int i = start_idx; i < gid; i++) {
            bv += fabsf(returns[i]) * fabsf(returns[i-1]);
        }
        
        // Adjust denominator for the actual number of terms used
        int actual_terms = gid - start_idx;
        if (actual_terms > 0) {
            bv *= pi_over_2 / actual_terms;
        } else {
            bv = 0.01f;  // Small default value to avoid division by zero
        }
        
        local_vol[gid] = sqrtf(bv);
        
        // Avoid division by zero
        float L = (local_vol[gid] > 0.0f) ? fabsf(returns[gid]) / local_vol[gid] : 0.0f;
        test_stats[gid] = L;
        
        // Debug output for first few threads
        if (gid < 5) {
            printf("Thread %d: return=%.6f, local_vol=%.6f, L=%.6f, threshold=%.2f\n", 
                   gid, returns[gid], local_vol[gid], L, threshold);
        }
        
        // Lee-Mykland critical value calculation
        // The threshold parameter is the critical value directly
        float critical_value = threshold;
        
        jump_flags[gid] = (L > critical_value) ? 1 : 0;  // Convert boolean to char (0 or 1)
        
        // Debug output for jumps detected
        if (L > critical_value && gid < 100) {
            printf("JUMP DETECTED at thread %d: L=%.6f > threshold=%.2f\n", gid, L, critical_value);
        }
    }
}

// Corrected BNS kernel - fixed tri-power quarticity scaling
__global__ void bns_kernel(
    const float* __restrict__ returns,
    float* __restrict__ rv,
    float* __restrict__ bv,
    float* __restrict__ tq,
    float* __restrict__ test_stats,
    const int n,
    const int window_size
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (gid < n - window_size) {
        float local_rv = 0.0f;
        float local_bv = 0.0f;
        float local_tq = 0.0f;
        
        const float pi_over_2 = 1.5707963267948966f;
        const float mu_43 = 1.67f;
        
        for (int i = 0; i < window_size; i++) {
            int idx = gid + i;
            float r = returns[idx];
            local_rv += r * r;
            
            if (i > 0) {
                local_bv += fabsf(r) * fabsf(returns[idx-1]);
            }
            
            if (i > 1) {
                float r1 = powf(fabsf(returns[idx]), 4.0f/3.0f);
                float r2 = powf(fabsf(returns[idx-1]), 4.0f/3.0f);
                float r3 = powf(fabsf(returns[idx-2]), 4.0f/3.0f);
                local_tq += r1 * r2 * r3;
            }
        }
        
        local_bv *= pi_over_2 * window_size / (window_size - 1);
        local_tq *= powf(mu_43, -3.0f) * window_size / (window_size - 2);
        
        rv[gid] = local_rv;
        bv[gid] = local_bv;
        tq[gid] = local_tq;
        
        if (local_rv > 0 && local_bv > 0) {
            float jump_component = local_rv - local_bv;
            float theta = 0.609f;
            float denominator = sqrtf(theta * local_tq / (local_bv * local_bv));
            
            if (denominator > 0) {
                test_stats[gid] = sqrtf(float(window_size)) * (jump_component / local_rv) / denominator;
            } else {
                test_stats[gid] = 0.0f;
            }
        } else {
            test_stats[gid] = 0.0f;
        }
    }
}

// Complete Hawkes process implementation with parameter estimation
__global__ void hawkes_intensity_kernel(
    const float* __restrict__ timestamps,
    float* __restrict__ intensity,
    float* __restrict__ branching_ratio,
    float* __restrict__ endogeneity,
    const int n,
    const float mu,
    const float phi,
    const float kappa
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (gid < n) {
        float t_current = timestamps[gid] * 1e-9f;
        float lambda = mu;
        
        for (int j = 0; j < gid; j++) {
            float t_j = timestamps[j] * 1e-9f;
            float time_diff = t_current - t_j;
            if (time_diff > 0) {
                lambda += phi * expf(-kappa * time_diff);
            }
        }
        
        intensity[gid] = lambda;
        
        float n_branch = phi / kappa;
        branching_ratio[gid] = n_branch;
        
        if (lambda > 0) {
            endogeneity[gid] = 1.0f - mu / lambda;
        } else {
            endogeneity[gid] = 0.0f;
        }
    }
}

// Poisson baseline model for trade arrival
__global__ void poisson_intensity_kernel(
    const float* __restrict__ timestamps,
    const float* __restrict__ prices,
    float* __restrict__ intensity,
    float* __restrict__ surprise_score,
    const int n,
    const int window_size,
    const float time_window_seconds
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (gid >= window_size && gid < n) {
        float t_current = timestamps[gid] * 1e-9f;
        float t_start = timestamps[gid - window_size] * 1e-9f;
        float duration = t_current - t_start;
        
        int count = 0;
        float window_start = t_current - time_window_seconds;
        
        for (int i = gid - window_size; i < gid; i++) {
            float t = timestamps[i] * 1e-9f;
            if (t >= window_start && t < t_current) {
                count++;
            }
        }
        
        float lambda_baseline = float(window_size) / duration;
        intensity[gid] = lambda_baseline;
        
        float expected = lambda_baseline * time_window_seconds;
        
        if (expected > 0) {
            surprise_score[gid] = (count - expected) / sqrtf(expected);
        } else {
            surprise_score[gid] = 0.0f;
        }
    }
}

// Composite burst detection combining Hawkes and Poisson
__global__ void burst_detection_kernel(
    const float* __restrict__ hawkes_intensity,
    const float* __restrict__ poisson_surprise,
    const float* __restrict__ branching_ratio,
    char* __restrict__ burst_flags,  // Changed bool* to char*
    float* __restrict__ burst_scores,
    const int n,
    const float hawkes_threshold,
    const float poisson_threshold,
    const float composite_threshold
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (gid < n) {
        float hawkes_score = branching_ratio[gid] / 0.8f;
        float poisson_score = fabsf(poisson_surprise[gid]) / 3.0f;
        
        float composite = 0.6f * hawkes_score + 0.4f * poisson_score;
        burst_scores[gid] = composite;
        
        burst_flags[gid] = ((branching_ratio[gid] > hawkes_threshold) ||
                           (fabsf(poisson_surprise[gid]) > poisson_threshold) ||
                           (composite > composite_threshold)) ? 1 : 0;  // Convert boolean to char (0 or 1)
    }
}

// Standardized returns kernel
__global__ void standardized_returns_kernel(
    const float* __restrict__ returns,
    const float* __restrict__ sigma,
    float* __restrict__ z_scores,
    const int n
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = gid; i < n; i += stride) {
        if (sigma[i] > 0) {
            z_scores[i] = returns[i] / sigma[i];
        } else {
            z_scores[i] = 0.0f;
        }
    }
}

// ==================== LAUNCHER IMPLEMENTATIONS ====================

void launch_garch_estimation(float* returns, float* sigma, int n, 
                             float omega, float alpha, float beta,
                             hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    size_t shared_mem_size = BLOCK_SIZE * sizeof(float);
    
    garch_kernel<<<blocks, BLOCK_SIZE, shared_mem_size, stream>>>(
        returns, sigma, n, omega, alpha, beta
    );
}

void launch_jump_detection(float* returns, float* local_vol, 
                           char* jump_flags, int n, float threshold,  // Changed bool* to char*
                           hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    float* test_stats;
    hipMalloc(&test_stats, n * sizeof(float));
    
    lee_mykland_kernel<<<blocks, BLOCK_SIZE, 0, stream>>>(
        returns, local_vol, test_stats, jump_flags, n, 100, threshold
    );
    
    hipFree(test_stats);
}

void launch_bns_computation(float* returns, float* rv, float* bv, float* tq,
                            float* stats, int n, int window, 
                            hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    bns_kernel<<<blocks, BLOCK_SIZE, 0, stream>>>(
        returns, rv, bv, tq, stats, n, window
    );
}

void launch_lee_mykland_computation(float* returns, float* local_vol, float* test_stats,
                                    char* jump_flags, int n, int window, float threshold,  // Changed bool* to char*
                                    hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    lee_mykland_kernel<<<blocks, BLOCK_SIZE, 0, stream>>>(
        returns, local_vol, test_stats, jump_flags, n, window, threshold
    );
    
    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Lee-Mykland kernel launch failed: %s\n", hipGetErrorString(err));
    }
    
    // Wait for kernel to complete and check for execution errors
    err = hipStreamSynchronize(stream);
    if (err != hipSuccess) {
        printf("Lee-Mykland kernel execution failed: %s\n", hipGetErrorString(err));
    }
}

void launch_hawkes_computation(float* timestamps, float* intensity, float* branching,
                               float* endogeneity, int n, float mu, float phi, float kappa,
                               hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hawkes_intensity_kernel<<<blocks, BLOCK_SIZE, 0, stream>>>(
        timestamps, intensity, branching, endogeneity, n, mu, phi, kappa
    );
}

void launch_poisson_computation(float* timestamps, float* prices, float* intensity,
                                float* surprise, int n, int window, float time_window,
                                hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    poisson_intensity_kernel<<<blocks, BLOCK_SIZE, 0, stream>>>(
        timestamps, prices, intensity, surprise, n, window, time_window
    );
}

void launch_burst_computation(float* hawkes_int, float* poisson_surp, float* branching,
                              char* flags, float* scores, int n, float h_thresh,  // Changed bool* to char*
                              float p_thresh, float c_thresh, 
                              hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    burst_detection_kernel<<<blocks, BLOCK_SIZE, 0, stream>>>(
        hawkes_int, poisson_surp, branching, flags, scores, n, h_thresh, p_thresh, c_thresh
    );
}

void launch_standardized_returns(float* returns, float* sigma, float* z_scores, int n,
                                 hipStream_t stream) {
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    standardized_returns_kernel<<<blocks, BLOCK_SIZE, 0, stream>>>(
        returns, sigma, z_scores, n
    );
}

}} // namespace surprise_metrics::cuda

